#include "hip/hip_runtime.h"
//%%cuda --name /content/src/direct_convolution.cu

/*including the required library*/
#include "direct_conv.h"
using namespace std;

__global__ void pad_input(float* f_in, float* f_out, int H, int W, int D, int pad)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int dep = blockIdx.z*blockDim.z+threadIdx.z;

    int new_H = H+2*pad; int new_W = W+2*pad; 
 
    int i = dep * new_H * new_W + col * new_W + row;
    int j = dep * H * W + (col - pad) *W+ (row - pad);

    if(col < new_H && row < new_W && dep < D)
    {
        if((col < pad || col > H+pad-1) || (row < pad || row > W+pad-1)) f_out[i] = 0;
        else f_out[i] = f_in[j];
    }
}

/*parallelization code */
__global__ 
void direct_convolution(int input_channels, int input_height, int input_width, int out_channels, int kernel_height,int kernel_width, 
                        int padding, int stride, int H_out, int W_out, int W_grid, int tile_w, float* X, float* W_filter, float* Y) {
  int n , m , h , w , c , p , q;
  n = blockIdx.x;
  m = blockIdx.y;
  h = (blockIdx.z / W_grid)*tile_w + threadIdx.y;
  w = (blockIdx.z % W_grid)*tile_w + threadIdx.x;

  input_height = input_height+padding;
  input_width = input_width+padding;

  if(h<H_out && w<W_out) {
    float temp=0;
    for(c = 0; c < input_channels; c++) {
      for(p = 0; p < kernel_height; p++) {
        for(q = 0; q < kernel_width; q++) {
          temp = temp + X[ n*(input_channels*input_height*input_width) + c*(input_height*input_width) + (h*stride+p)*(input_width) + (w*stride+q)] 
                          * W_filter[ m*(input_channels*kernel_height*kernel_width) + c*(kernel_height*kernel_width) + p*(kernel_height) + q];
        }
      }
    }
    Y[n*(out_channels*H_out*W_out) + m*(H_out*W_out) + h*(W_out) + w] = temp;
  }
}

/*forward pass function declared in direc_conv.hpp library*/
float* Direct::passforward(int out_channels, int input_channels, int kernel_height, int kernel_width, int padding, int stride, 
                          float* d_weights,int batchsize_of_data, int input_height, int input_width, float* d_input) {
  if(kernel_height > input_height || kernel_width > input_width){
    cout << "kernel size is too big " << endl;
    exit(EXIT_FAILURE);
  }
  
  hipError_t err = hipSuccess;
  
  /* The rest of the code assumes that padding = x means x/2 on either ends hence the modification */
  padding = 2*padding;

  /* size of matrix with padding*/ 
  int size_input_matrix = batchsize_of_data * input_channels * (input_height+padding) * (input_width+padding) * sizeof(float);   // size of input matrix after padding

  /* calculating size of output matrix*/
  int H_out = (input_height - kernel_height + padding + stride)/stride;
  int W_out = (input_width - kernel_width + padding + stride)/stride;
  int size_output_matrix = batchsize_of_data * out_channels * H_out * W_out * sizeof(float);
  
  /*allocating memory for input  matrix with padding*/
  float *h_X = (float*)malloc(size_input_matrix);  
 
  /* memory allocation check*/
  if (h_X == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  /* Padding */
  int new_input_height = input_height + padding;
  int new_input_width = input_width + padding;

  float *pad_input_in = NULL; 
  hipMalloc((void **)&pad_input_in, input_height * input_width * input_channels * sizeof(float));
  
  float *pad_input_out = NULL; 
  hipMalloc((void **)&pad_input_out, new_input_height * new_input_width * input_channels * sizeof(float));

  dim3 threads1(1, 1, 1);
  dim3 grid1(new_input_height, new_input_width, input_channels);

  for(int i = 0; i < batchsize_of_data; i++)
  {
    hipMemcpy(pad_input_in, &d_input[i * input_channels * input_height * input_width],
              input_height * input_width * input_channels * sizeof(float) , hipMemcpyHostToDevice);
    
    pad_input<<<grid1,threads1>>>(pad_input_in, pad_input_out, input_height, input_width, input_channels, padding/2);
    err = hipGetLastError(); 
    if(err!=hipSuccess) {
      fprintf(stderr, "Failed to launch pad input (error code %s)!\n", hipGetErrorString(err)); 
      exit(EXIT_FAILURE);
    }
    
    hipMemcpy(&h_X[i * input_channels * new_input_height * new_input_width], pad_input_out,
              new_input_height * new_input_width * input_channels * sizeof(float), hipMemcpyDeviceToHost);
  }
  hipFree(pad_input_in); 
  hipFree(pad_input_out);

  float *d_X, *d_Y, *d_W; 

  /*allocating memory for padded matrix in the device*/
  err = hipMalloc((void**)&d_X, size_input_matrix);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector d_X (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  /* copying padded matrix to device */
  err = hipMemcpy(d_X , h_X , size_input_matrix , hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector h_X from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
 
  /* Renaming the kernel weights pointer (input is in device memory) */
  d_W = d_weights; 

  /*allocating memory for the output matrix*/
  err = hipMalloc((void**)&d_Y, size_output_matrix);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector d_Y (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  /* making sure that 1024 threads isn't crossed*/
  int tile_width = 2 , tile_height = 2;   
  int w_grid = ceil((W_out*1.0) / tile_width);
  int h_grid = ceil((H_out*1.0) / tile_height);
 
  int temp  = w_grid * h_grid;
  dim3 grid(batchsize_of_data , out_channels , temp);
  dim3 block(tile_width , tile_height , 1);
 
  /* calling the direct_convolution kernel */  
  direct_convolution<<< grid, block >>>(input_channels, input_height, input_width, out_channels, kernel_height, kernel_width, 
                                        padding, stride, H_out, W_out, w_grid, tile_width, d_X, d_W, d_Y);

  err = hipGetLastError();

  /*checking if the device program is executed or not*/
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch reduce1 kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
 
  /* releasing all the device and host vectors */
  err = hipFree(d_X);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector X (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
 
  /*releasing the memory*/
  free(h_X);

  /*Return the CUDA Array*/
  return d_Y;      
}